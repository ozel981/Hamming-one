#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""


#include "GPUHammingOne.cuh"

/*__device__ bool IsHammingOne(std::string s1, std::string s2)
{
	int count = 0;
	for (int i = 0; i < s1.length(); i++)
	{
		if (s1[i] != s2[i]) count++;
		if (count > 1) return false;
	}
	return count == 1;
}
*/
__global__ void CalculateHammingOne(int* count, Data* d_data)
{
	int index = (blockIdx.x * 1024) + threadIdx.x;

	if (index < d_data->count)
	{
		count[index] = d_data->set[index][1];
		/*for (int i = index + 1; i < d_data->count; i++)
		{
			//if (IsHammingOne(set[index], set[i])) counts[index]++;
			int differencesCount = 0;
			for (int j = 0; j < d_data->length; j++)
			{
				if (d_data->set[index][j] != d_data->set[i][j]) differencesCount++;
				if (differencesCount > 1) break;
			}
			if (differencesCount == 1) count[index]+=1;
		}*/
	}
}


extern "C" int GPUHammingOneCount(Data* h_data)
{
	Data* d_data;
	int n = h_data->count;
	int* d_count;
	int* h_count = new int[n];
	hipMalloc((void**)&d_data, sizeof(Data));
	hipMalloc((void**)&d_count, n * sizeof(int));
	hipMemcpy(d_data, h_data, sizeof(Data), hipMemcpyHostToDevice);
	CalculateHammingOne << <1 + (n / 1024), 1024 >> > (d_count, d_data);
	hipMemcpy(h_count, d_count, n * sizeof(int), hipMemcpyDeviceToHost);

	int count = 0;

	for (int i = 0; i < n; i++)
	{
		count += h_count[i];
	}

	hipFree(d_count);
	hipFree(d_data);
	return count;
}
