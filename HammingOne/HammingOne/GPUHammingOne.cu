#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <stdio.h>

#define INTERVAL_LENGTH 5

#include "GPUHammingOne.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

__global__ void CalculateHammingOne(int* val, bool* set, int from, int n, int l, int radius)
{
	int index = (blockIdx.x * 1000) + threadIdx.x + (10000 * radius);
	if (index < n)
	{
		int differencesCount;
		for (int i = from > index + 1 ? from : index + 1; i < n; i++)
		{
			differencesCount = 0;
			for (int j = 0; j < l; j++)
			{
				if (set[index * l + j] != set[i * l + j]) differencesCount++;
				if (differencesCount > 1) break;
			}
			if (differencesCount == 1)
			{
				atomicAdd(val, 1);
				printf("Hamming one distance: [%d]x[%d]\n ", index, i);
			}
		}
	}
}


extern "C" int GPUHammingOneCount(Data* h_data)
{
	//Data* d_data;
	int n = h_data->count;
	int l = h_data->length;
	bool* h_set = new bool[l*n];
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < l; j++)
		{
			h_set[i * l + j] = h_data->set[i][j];
		}
	}
	bool* d_set;
	hipMalloc((void**)&d_set, l * n * sizeof(bool));
	hipMemcpy(d_set, h_set, l * n * sizeof(bool), hipMemcpyHostToDevice);
	int h_val = 0;
	int* d_val;
	hipMalloc(&d_val, sizeof(int));
	hipMemcpy(d_val, &h_val, sizeof(int), hipMemcpyHostToDevice);
	for (int radius = 0; radius <= n / 10000; radius++)
	{
		for (int i = 0; i < INTERVAL_LENGTH; i++)
		{
			int from = i * (n / INTERVAL_LENGTH);
			int to = (i + 1)*(n / INTERVAL_LENGTH);
			CalculateHammingOne << <10, 1000 >> > (d_val, d_set, from, to, l, radius);
		}
	}
	
	hipMemcpy(&h_val, d_val, sizeof(int), hipMemcpyDeviceToHost);


	gpuErrchk(hipPeekAtLastError());
	

	hipFree(d_set);

	return h_val;
}
