#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>


#include "GPUHammingOne.cuh"

__global__ void CalculateHammingOne(int* count, bool* set, int n, int l)
{
	int index = (blockIdx.x * 1024) + threadIdx.x;
	if (index < (n + 1) / 2)
	{
		count[index] = 0;
		for (int i = index + 1; i < n; i++)
		{
			int differencesCount = 0;
			for (int j = 0; j < l; j++)
			{
				if (set[index * l + j] != set[i * l + j]) differencesCount++;
				if (differencesCount > 1) break;
			}
			if (differencesCount == 1)
			{
				count[index] += 1;
				printf("Hamming one distance: [%d]x[%d]\n", index, i);
			}
		}	
		if (n - 1 - index != index)
		{
			index = n - 1 - index;
			count[index] = 0;
			for (int i = index + 1; i < n; i++)
			{
				int differencesCount = 0;
				for (int j = 0; j < l; j++)
				{
					if (set[index * l + j] != set[i * l + j]) differencesCount++;
					if (differencesCount > 1) break;
				}
				if (differencesCount == 1)
				{
					count[index] += 1;
					printf("Hamming one distance: [%d]x[%d]\n", index, i);
				}
			}
		}
	}
}


extern "C" int GPUHammingOneCount(Data* h_data)
{
	//Data* d_data;
	int n = h_data->count;
	int l = h_data->length;
	bool* h_set = new bool[l*n];
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < l; j++)
		{
			h_set[i * l + j] = h_data->set[i][j];
		}
	}
	bool* d_set;
	hipMalloc((void**)&d_set, l * n * sizeof(bool));
	hipMemcpy(d_set, h_set, l * n * sizeof(bool), hipMemcpyHostToDevice);
	int* h_count = new int[n];
	int* d_count;
	hipMalloc((void**)&d_count, n * sizeof(int));
	int ndiv2 = n / 2;
	CalculateHammingOne << <1 + (ndiv2 / 1024), 1024 >> > (d_count, d_set, n, l);
	hipMemcpy(h_count, d_count, n * sizeof(int), hipMemcpyDeviceToHost);

	int count = 0;

	hipFree(d_count);
	hipFree(d_set);
	return thrust::reduce(thrust::host, h_count, h_count + n, 0);
}
