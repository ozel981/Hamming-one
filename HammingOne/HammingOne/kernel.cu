
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "DataGenerator.h";

int main()
{
	int l, n, x;
	printf("Find the number of vectors pairs with Hamming one distance by \n 1.CPU \n 2.GPU \n 3.Generate test data\n:");
	scanf("%d", &x);
	if (x == 1)
	{
		printf("Sory no CPU version. \n");
	}
	if (x == 2)
	{
		printf("Sory no GPU version. \n");
	}
	if (x == 3)
	{
		printf("Enter set of vectors count: ");
		scanf("%d", &n);
		printf("Enter vector length: ");
		scanf("%d", &l);
		GenerateData("test2", n, l);
	}

    return 0;
}

