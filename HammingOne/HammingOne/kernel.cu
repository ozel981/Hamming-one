
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "DataGenerator.h"
#include "CPUHammingOne.h"
#include "GPUHammingOne.cuh"
#include "ReadData.h"

extern "C" int GPUHammingOneCount(Data* data);

int main()
{
	int l, n, x;
	printf("Find the number of vectors pairs with Hamming one distance by \n 1.CPU \n 2.GPU \n 3.Generate test data\n:");
	scanf("%d", &x);
	if (x == 1)
	{
		int count = CPUHammingOneCount(&ReadData("test3"));
		printf("Hamming one distance count: %d \n", count);
	}
	if (x == 2)
	{
		int count = GPUHammingOneCount(&ReadData("test3"));
		std::cout << count << std::endl;
		printf("Hamming one distance count: %d \n", count);
	}
	if (x == 3)
	{
		printf("Enter set of vectors count: ");
		scanf("%d", &n);
		printf("Enter vector length: ");
		scanf("%d", &l);
		GenerateData("test3", n, l);
	}

    return 0;
}

